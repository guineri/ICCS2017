
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void calculate(int *d_buffer){
	int ix = blockIdx.x*blockDim.x + threadIdx.x;

	d_buffer[ix] += d_buffer[ix];


	__syncthreads();


}


__global__ void compute1(int *d_buffer){
	int ix = blockIdx.x*blockDim.x + threadIdx.x;

	d_buffer[ix] = d_buffer[ix]*2;


	__syncthreads();


}

__global__ void compute2(int *d_buffer){
	int ix = blockIdx.x*blockDim.x + threadIdx.x;

	d_buffer[ix] = d_buffer[ix]*10;


	__syncthreads();


}

int main(int argc, char *argv[]){


	int *matriz_1,*matriz_2,*h_matriz;

	int N,i,j,iter;

	int loop;

	//dimensão da matriz
	N = atoi(argv[1]);
	//numero de iterações
	iter = atoi(argv[2]);
	
	loop = atoi(argv[3]);	
	
	h_matriz = (int*) malloc(sizeof(int)*N*N);

	hipMalloc(&matriz_1,sizeof(int)*N*N);

	for(i=0;i<N;i++){
		for(j=0;j<N;j++){
			h_matriz[i*N+j] = (i*N+j)+1;
		}
	}

	hipMemcpy(matriz_1,h_matriz,N*N*sizeof(int),hipMemcpyHostToDevice);

	dim3 grid, block;
	block.x = 1024;
	grid.x = (N + block.x - 1) / block.x;

	for(i=0;i<iter;i++){
		for(j=0;j<loop;j++){
			calculate<<<grid,block>>>(matriz_1);
		}
		for(j=0;j<loop;j++){
			compute1<<<grid,block>>>(matriz_1);
		}
		for(j=0;j<loop;j++){
			compute2<<<grid,block>>>(matriz_1);
		}
		

	}


	return 0;
}











